#include <hip/hip_runtime.h>


#include <stdio.h>


#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 200)
  # error printf is only supported on devices of compute capability 2.0 and higher, please compile with -arch=sm_20 or higher
#endif

extern "C"
void runCudaPart();


__global__ void helloCUDA()
{
    int tid = blockIdx.x;
    printf("Hello block %d thread %d \n",tid , threadIdx.x );
}

extern "C"
void runCudaPart()
{
    // all your cuda code here
    helloCUDA<<<5,1>>>();
    hipDeviceSynchronize();
}
